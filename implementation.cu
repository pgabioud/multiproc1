#include "hip/hip_runtime.h"
/*
============================================================================
Filename    : algorithm.c
Author      : Your name goes here
SCIPER      : Your SCIPER number
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>
using namespace std;

// CPU Baseline
void array_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for(int n=0; n<(int) iterations; n++)
    {
        for(int i=1; i<length-1; i++)
        {
            for(int j=1; j<length-1; j++)
            {
                output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
                                            input[(i-1)*(length)+(j)]   +
                                            input[(i-1)*(length)+(j+1)] +
                                            input[(i)*(length)+(j-1)]   +
                                            input[(i)*(length)+(j)]     +
                                            input[(i)*(length)+(j+1)]   +
                                            input[(i+1)*(length)+(j-1)] +
                                            input[(i+1)*(length)+(j)]   +
                                            input[(i+1)*(length)+(j+1)] ) / 9;

            }
        }
        output[(length/2-1)*length+(length/2-1)] = 1000;
        output[(length/2)*length+(length/2-1)]   = 1000;
        output[(length/2-1)*length+(length/2)]   = 1000;
        output[(length/2)*length+(length/2)]     = 1000;

        temp = input;
        input = output;
        output = temp;
    }
}

__global__ void GPU_processing(double *input, double *output, int length) {
    //for(int i = 0; i<iterations; i++) {
        int x = (blockIdx.x*blockDim.x) + threadIdx.x;
        int y = (blockIdx.y*blockDim.y) + threadIdx.y;
        int element_id = (y*length) + x;

        if (x >= length || y >= length || x%(length-1) == 0 || y%(length-1) == 0 ||
            (x==length/2 - 1 && (y==length/2 || y==length/2-1)) ||
            (x==length/2 && (y==length/2 || y==length/2-1))) return;

        output[element_id] = (input[(y-1)*(length)+(x-1)] +
                                            input[(y-1)*(length)+(x)]   +
                                            input[(y-1)*(length)+(x+1)] +
                                            input[(y)*(length)+(x-1)]   +
                                            input[(y)*(length)+(x)]     +
                                            input[(y)*(length)+(x+1)]   +
                                            input[(y+1)*(length)+(x-1)] +
                                            input[(y+1)*(length)+(x)]   +
                                            input[(y+1)*(length)+(x+1)] ) / 9;
    //}
}

// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
    //Cuda events for calculating elapsed time
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);

    /* Preprocessing goes here */
    double* gpu_output;
    hipMalloc((void**)&gpu_output, length*length*sizeof(double));
    double* gpu_input;
    hipMalloc((void**)&gpu_input, length*length*sizeof(double));
    double* temp;

    hipEventRecord(cpy_H2D_start);

    /* Copying array from host to device goes here */
    hipMemcpy((void*)gpu_input, (void*)input, length*length*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void*)gpu_output, (void*)output, length*length*sizeof(double), hipMemcpyHostToDevice);

    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);

    //Copy array from host to device
    hipEventRecord(comp_start);

    /* GPU calculation goes here */
    dim3 thrsPerBlock(16,16);//256 threads par blocks
    int nbTB = ceil(sqrt(ceil(length*length/256)));
    dim3 nBlks(nbTB, nbTB);
	
	
    for(int i = 0; i < iterations; i++) {
        GPU_processing<<< nBlks, thrsPerBlock>>>(gpu_input, gpu_output, length);
        hipDeviceSynchronize();
        temp = gpu_input;
        gpu_input = gpu_output;
        gpu_output = temp;
    }


    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);

    hipEventRecord(cpy_D2H_start);

    /* Copying array from device to host goes here */
    hipMemcpy((void*)output, (void*)gpu_input, length*length*sizeof(double), hipMemcpyDeviceToHost);

    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);

    /* Postprocessing goes here */
    hipFree(gpu_input);
    hipFree(gpu_output);


    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout<<"Host to Device MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout<<"Computation takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout<<"Device to Host MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;
}
